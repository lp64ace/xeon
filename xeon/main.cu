#include <gpu/texture.h>
#include <gpu/shader.h>
#include <glad/glad.h>
#include <gpu/compute.h>
#include <wnd/wndlist.h>

const int SCREEN_WIDTH = 800;
const int SCREEN_HEIGHT = 480;
int tx = 32;
int ty = 32;

WND* wnd;

int main ( )
{
	init_api();

	wnd = new WND();
	wnd->Create(NULL, "Xeon", Box(CW_USEDEFAULT, CW_USEDEFAULT, SCREEN_WIDTH, SCREEN_HEIGHT));
	wnd->Show();

	wnd->CreateContext();
	APP->setFpsCap(60.0f);

	Texture tex(SCREEN_WIDTH, SCREEN_HEIGHT, false);
	launch_kernel(tex, 10, tx, ty);

	Shader shader;
	shader.load ( "shaders/basic_vertex.vert" , ShaderType::Vertex );
	shader.load ( "shaders/basic_fragment.frag" , ShaderType::Fragment );
	shader.build ( );
	
	float quad_data [ 24 ] =
	{
		-1.f,  1.f, 0.0f, 1.0f,
		 1.f,  1.f, 1.0f, 1.0f,
		-1.f, -1.f, 0.0f, 0.0f,

		-1.f, -1.f, 0.0f, 0.0f,
		 1.f, -1.f, 1.0f, 0.0f,
		 1.f,  1.f, 1.0f, 1.0f
	};

	unsigned int VBO , VAO;
	glGenVertexArrays ( 1 , &VAO );
	glGenBuffers ( 1 , &VBO );
	glBindVertexArray ( VAO );

	glBindBuffer ( GL_ARRAY_BUFFER , VBO );
	glBufferData ( GL_ARRAY_BUFFER , sizeof ( quad_data ) , quad_data , GL_STATIC_DRAW );

	glVertexAttribPointer ( 0 , 2 , GL_FLOAT , GL_FALSE , 4 * 4 , ( void * ) 0 );
	glEnableVertexAttribArray ( 0 );

	glVertexAttribPointer ( 1 , 2 , GL_FLOAT , GL_FALSE , 4 * 4 , ( void * ) 8 );
	glEnableVertexAttribArray ( 1 );
	
	while (APP->PollWindowsEvents()) {

		glClearColor(0.0f, 0.2f, 0.4f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		shader.bind();
		tex.bind();
		tex.upload_to_gpu();

		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLES, 0, 6);
		
		APP->appSwapBuffers();
	}
	
	checkCudaErrors(hipDeviceSynchronize());
	hipDeviceReset();
}
