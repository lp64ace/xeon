#include <gpu/texture.h>
#include <gpu/shader.h>
#include <glad/glad.h>
#include <gpu/compute.h>
//#include <wnd/wndlist.h>
#include <GLFW/glfw3.h>

const int SCREEN_WIDTH = 800;
const int SCREEN_HEIGHT = 480;
int tx = 8;
int ty = 8;

//WND* wnd;

Hitable** objects;
ShapeList** world;
Camera** d_camera;
hiprandState* d_rand_state;

void launch_kernel(Texture& screen_tex, int object_count, int block_x = 16, int block_y = 16)
{
	int width = screen_tex.twidth;
	int height = screen_tex.theight;

	/* Allocate world objects on the GPU. */
	checkCudaErrors(hipMalloc((void**)&objects, object_count * sizeof(Hitable*)));
	checkCudaErrors(hipMalloc((void**)&world, sizeof(ShapeList*)));
	checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));
	checkCudaErrors(hipMalloc((void**)&d_rand_state, width * height * sizeof(hiprandState)));

	dim3 blocks(width / block_x + 1, height / block_y + 1);
	dim3 threads(block_x, block_y);

	render_init<<<blocks, threads>>>(width, height, d_rand_state);
	checkCudaErrors(hipDeviceSynchronize());

	/* Init the world on the GPU. */
	create_world<<<1, 1>>>(objects, object_count, world, d_camera, d_rand_state, width, height);
	checkCudaErrors(hipDeviceSynchronize());

	int fb_size = 4 * screen_tex.twidth * screen_tex.theight;
	checkCudaErrors(hipMallocManaged((void**)&screen_tex.buffer, fb_size));

	/* Start the ray tracer. */
	render<<<blocks, threads>>>(screen_tex.buffer, width, height, 100, d_camera, world, d_rand_state);
	checkCudaErrors(hipDeviceSynchronize());
}

int main ( )
{
	/*init_api();

	wnd = new WND();
	wnd->Create(NULL, "Xeon", Box(CW_USEDEFAULT, CW_USEDEFAULT, SCREEN_WIDTH, SCREEN_HEIGHT));
	wnd->Show();

	wnd->CreateContext();
	APP->setFpsCap(60.0f);*/
	
	glfwInit();
	GLFWwindow* window = glfwCreateWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "Xeon", NULL, NULL);
	glfwMakeContextCurrent(window);

	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		printf("[OpenGL] glfwGetProcAddress error!\n");
	}

	Texture tex(SCREEN_WIDTH, SCREEN_HEIGHT, false);
	launch_kernel(tex, 4, tx, ty);

	Shader shader;
	shader.load ( "shaders/basic_vertex.vert" , ShaderType::Vertex );
	shader.load ( "shaders/basic_fragment.frag" , ShaderType::Fragment );
	shader.build ( );
	
	float quad_data [ 24 ] =
	{
		-1.f,  1.f, 0.0f, 1.0f,
		 1.f,  1.f, 1.0f, 1.0f,
		-1.f, -1.f, 0.0f, 0.0f,

		-1.f, -1.f, 0.0f, 0.0f,
		 1.f, -1.f, 1.0f, 0.0f,
		 1.f,  1.f, 1.0f, 1.0f
	};

	unsigned int VBO , VAO;
	glGenVertexArrays ( 1 , &VAO );
	glGenBuffers ( 1 , &VBO );
	glBindVertexArray ( VAO );

	glBindBuffer ( GL_ARRAY_BUFFER , VBO );
	glBufferData ( GL_ARRAY_BUFFER , sizeof ( quad_data ) , quad_data , GL_STATIC_DRAW );

	glVertexAttribPointer ( 0 , 2 , GL_FLOAT , GL_FALSE , 4 * 4 , ( void * ) 0 );
	glEnableVertexAttribArray ( 0 );

	glVertexAttribPointer ( 1 , 2 , GL_FLOAT , GL_FALSE , 4 * 4 , ( void * ) 8 );
	glEnableVertexAttribArray ( 1 );
	
	while (/*APP->PollWindowsEvents()*/!glfwWindowShouldClose(window)) {

		glfwPollEvents();

		glClearColor(0.0f, 0.2f, 0.4f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT);

		shader.bind();
		tex.bind();
		tex.upload_to_gpu();

		glBindVertexArray(VAO);
		glDrawArrays(GL_TRIANGLES, 0, 6);
		
		//APP->appSwapBuffers();
		glfwSwapBuffers(window);
	}
	
	checkCudaErrors(hipDeviceSynchronize());
	free_world<<<1, 1>>>(objects, world, d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(world));
	checkCudaErrors(hipFree(objects));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(tex.buffer));
	checkCudaErrors(hipDeviceReset());
}
